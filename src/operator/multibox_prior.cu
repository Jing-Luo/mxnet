#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2016 by Contributors
 * \file multibox_prior.cu
 * \brief generate multibox prior boxes cuda kernels
 * \author Joshua Zhang
*/

#include "./multibox_prior-inl.h"

#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

#define MULTIBOXPRIOR_CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    CHECK_EQ(error, hipSuccess) << " " << hipGetErrorString(error); \
  } while (0)

namespace mshadow {
namespace cuda {
template<typename DType>
__global__ void AssignPriors(DType *out, float size, float sqrt_ratio, int in_width,
                             int in_height, float step_x, float step_y, int stride, int offset) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= in_width * in_height) return;
  int r = index / in_width;
  int c = index % in_width;
  float center_x = (c + 0.5) * step_x;
  float center_y = (r + 0.5) * step_y;
  float w = size * sqrt_ratio / 2;  // half width
  float h = size / sqrt_ratio / 2;  // half height
  DType *ptr = out + index * stride + 4 * offset;
  *(ptr++) = center_x - w;  // xmin
  *(ptr++) = center_y - h;  // ymin
  *(ptr++) = center_x + w;  // xmax
  *(ptr++) = center_y + h;  // ymax
}
}  // namespace cuda

template<typename DType>
inline void MultiBoxPriorForward(const Tensor<gpu, 2, DType> &out,
                            const std::vector<float> &sizes,
                            const std::vector<float> &ratios,
                            int in_width, int in_height) {
  CHECK_EQ(out.CheckContiguous(), true);
  hipStream_t stream = Stream<gpu>::GetStream(out.stream_);
  DType *out_ptr = out.dptr_;
  float step_x = 1.f / in_width;
  float step_y = 1.f / in_height;
  int num_sizes = static_cast<int>(sizes.size());
  int num_ratios = static_cast<int>(ratios.size());

  int num_thread = THREADS_PER_WARP * WARPS_PER_BLOCK;
  dim3 thread_dim(num_thread);
  dim3 block_dim((in_width * in_height - 1) / num_thread + 1);

  int stride = 4 * (num_sizes + num_ratios - 1);
  int offset = 0;
  // ratio = 1, various sizes
  for (int i = 0; i < num_sizes; ++i) {
    cuda::AssignPriors<DType><<<block_dim, thread_dim, 0, stream>>>(out_ptr,
      sizes[i], 1.f, in_width, in_height, step_x, step_y, stride, offset);
    ++offset;
  }
  MULTIBOXPRIOR_CUDA_CHECK(hipPeekAtLastError());

  // size = sizes[0], various ratios
  for (int j = 1; j < num_ratios; ++j) {
    cuda::AssignPriors<DType><<<block_dim, thread_dim, 0, stream>>>(out_ptr,
      sizes[0], sqrtf(ratios[j]), in_width, in_height, step_x, step_y, stride, offset);
    ++offset;
  }
  MULTIBOXPRIOR_CUDA_CHECK(hipPeekAtLastError());
}
}  // namespace mshadow

namespace mxnet {
namespace op {
template<>
Operator* CreateOp<gpu>(MultiBoxPriorParam param, int dtype) {
  Operator *op = NULL;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new MultiBoxPriorOp<gpu, DType>(param);
  });
  return op;
}

}  // namespace op
}  // namespace mxnet
