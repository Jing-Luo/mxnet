#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2016 by Contributors
 * \file gird_anchor.cc
 * \brief generate grid anchors cuda impl
 * \author Joshua Zhang
*/

#include "./grid_anchor-inl.h"

#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

#define GRIDANCHOR_CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    CHECK_EQ(error, hipSuccess) << " " << hipGetErrorString(error); \
  } while (0)

namespace mshadow {
namespace cuda {
template<typename DType>
__global__ void AssignAnchors(DType *out, int in_width, int in_height,
                              float step_x, float step_y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= in_width * in_height) return;
  int r = index / in_width;
  int c = index % in_width;
  float center_x = (c + 0.5) * step_x;
  float center_y = (r + 0.5) * step_y;
  DType *ptr = out + index;
  *ptr = center_x;
  ptr += in_width * in_height;
  *ptr = center_y;
}

template<typename DType>
__global__ void PrintOutput(DType *ptr, int num) {
  for (int i = 0; i < num; ++i) {
    printf("%d: %f, ", i, float(ptr[i]));
  }
}
}  // namespace cuda

template<typename DType>
inline void GridAnchorForward(const Tensor<gpu, 3, DType> &out,
                              int in_width, int in_height) {
  CHECK_EQ(out.CheckContiguous(), true);
  hipStream_t stream = Stream<gpu>::GetStream(out.stream_);
  DType *out_ptr = out.dptr_;
  float step_x = 1.f / in_width;
  float step_y = 1.f / in_height;

  int num_thread = THREADS_PER_WARP * WARPS_PER_BLOCK;
  dim3 thread_dim(num_thread);
  dim3 block_dim((in_width * in_height - 1) / num_thread + 1);

  cuda::AssignAnchors<DType><<<block_dim, thread_dim, 0, stream>>>(out_ptr,
    in_width, in_height, step_x, step_y);
  GRIDANCHOR_CUDA_CHECK(hipPeekAtLastError());

  // cuda::PrintOutput<DType><<<1,1>>>(out.dptr_, in_width * in_height);
  // LOG(INFO) << "Y:";
  // cuda::PrintOutput<DType><<<1,1>>>(out.dptr_ + in_width * in_height,
  //   in_width * in_height);
}
}  // namespace mshadow

namespace mxnet {
namespace op {
template<>
Operator* CreateOp<gpu>(GridAnchorParam param, int dtype) {
  Operator *op = NULL;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new GridAnchorOp<gpu, DType>(param);
  });
  return op;
}

}  // namespace op
}  // namespace mxnet
