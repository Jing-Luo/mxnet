#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2016 by Contributors
 * \file grid_anchor_detection-inl.h
 * \brief post-process grid anchor predictions cuda impl
 * \author Joshua Zhang
*/
#include "./grid_anchor_detection-inl.h"

#define WARPS_PER_BLOCK 16
#define THREADS_PER_WARP 32

#define GRID_ANCHOR_DETECTION_CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    CHECK_EQ(error, hipSuccess) << " " << hipGetErrorString(error); \
  } while (0)

namespace mshadow {
namespace cuda {
template<typename DType>
__device__ void Clip(DType *value, DType lower, DType upper) {
  if ((*value) < lower) *value = lower;
  if ((*value) > upper) *value = upper;
}

template<typename DType>
__global__ void MergePredictions(DType *out, const DType *cls_prob,
                                 const DType *box_pred, const DType *anchors,
                                 int num_classes, int num_spatial,
                                 int num_batches, float threshold, bool clip) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= num_batches * num_spatial) return;
  for (int i = index; i < num_batches * num_spatial; i += blockDim.x * gridDim.x) {
    int n_batch = i / num_spatial;
    int n_anchor = i % num_spatial;
    const DType *p_cls_prob = cls_prob + n_batch * num_classes * num_spatial;
    const DType *p_box_pred = box_pred + n_batch * num_spatial * 4;
    DType *p_out = out + n_batch * num_spatial * 6;
    DType score = -1;
    int id = 0;
    for (int j = 1; j < num_classes; ++j) {
      DType temp = p_cls_prob[j * num_spatial + n_anchor];
      if (temp > score) {
        score = temp;
        id = j;
      }
    }
    if (id > 0 && score < threshold) {
      id = 0;
    }
    p_out[n_anchor * 6] = id - 1;  // restore original class id
    p_out[n_anchor * 6 + 1] = (id == 0 ? DType(-1) : score);
    DType center_x = anchors[n_anchor];
    DType center_y = anchors[n_anchor + num_spatial];
    DType x = center_x + p_box_pred[n_anchor];
    DType y = center_y + p_box_pred[n_anchor + num_spatial];
    DType width = pow(p_box_pred[n_anchor + num_spatial * 2], 2) / 2;
    DType height = pow(p_box_pred[n_anchor + num_spatial * 3], 2) / 2;
    // printf("%f, %f, %f, %f, %f\n", float(center_x), float(center_y), float(width),
    //   float(height), float(score));
    DType xmin = x - width;
    DType ymin = y - height;
    DType xmax = x + width;
    DType ymax = y + height;
    if (clip) {
      Clip(&xmin, DType(0), DType(1));
      Clip(&ymin, DType(0), DType(1));
      Clip(&xmax, DType(0), DType(1));
      Clip(&ymax, DType(0), DType(1));
    }
    p_out[n_anchor * 6 + 2] = xmin;
    p_out[n_anchor * 6 + 3] = ymin;
    p_out[n_anchor * 6 + 4] = xmax;
    p_out[n_anchor * 6 + 5] = ymax;
  }
}

template<typename DType>
__global__ void MergeSortDescend(DType *src, DType *dst, int size,
                                 int width, int slices, int step, int offset) {
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  int start = width * index * slices;
  for (int slice = 0; slice < slices; ++slice) {
    if (start >= size) break;
    int middle = start + (width >> 1);
    if (middle > size) middle = size;
    int end = start + width;
    if (end > size) end = size;
    int i = start;
    int j = middle;
    for (int k = start; k < end; ++k) {
      DType score_i = i < size ? src[i * step + offset] : DType(-1);
      DType score_j = j < size ? src[j * step + offset] : DType(-1);
      if (i < middle && (j >= end || score_i > score_j)) {
        for (int n = 0; n < step; ++n) {
          dst[k * step + n] = src[i * step + n];
        }
        ++i;
      } else {
        for (int n = 0; n < step; ++n) {
          dst[k * step + n] = src[j * step + n];
        }
        ++j;
      }
    }
    start += width;
  }
}

template<typename DType>
__device__ void CalculateOverlap(const DType *a, const DType *b, DType *iou) {
  DType w = max(DType(0), min(a[2], b[2]) - max(a[0], b[0]));
  DType h = max(DType(0), min(a[3], b[3]) - max(a[1], b[1]));
  DType i = w * h;
  DType u = (a[2] - a[0]) * (a[3] - a[1]) + (b[2] - b[0]) * (b[3] - b[1]) - i;
  (*iou) =  u <= 0.f ? static_cast<DType>(0) : static_cast<DType>(i / u);
}

template<typename DType>
__global__ void ApplyNMS(DType *out, int pos, int num_anchors,
                         int step, int id_index, int loc_index,
                         bool force_suppress, float nms_threshold) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  DType compare_id = out[pos * step + id_index];
  if (compare_id < 0) return;  // not a valid positive detection, skip
  DType *compare_loc_ptr = out + pos * step + loc_index;
  for (int i = index; i < num_anchors; i += blockDim.x * gridDim.x) {
    if (i <= pos) continue;
    DType class_id = out[i * step + id_index];
    if (class_id < 0) continue;
    if (force_suppress || (class_id == compare_id)) {
      DType iou;
      CalculateOverlap(compare_loc_ptr, out + i * step + loc_index, &iou);
      if (iou >= nms_threshold) {
        out[i * step + id_index] = -1;
      }
    }
  }
}
}  // namespace cuda

template<typename DType>
inline void GridAnchorDetectionForward(const Tensor<gpu, 3, DType> &out,
                                     const Tensor<gpu, 3, DType> &cls_prob,
                                     const Tensor<gpu, 3, DType> &box_pred,
                                     const Tensor<gpu, 3, DType> &anchors,
                                     float threshold, bool clip) {
  int num_classes = cls_prob.size(1);
  int num_spatial = cls_prob.size(2);
  int num_batches = cls_prob.size(0);
  const int num_threads = THREADS_PER_WARP * WARPS_PER_BLOCK;
  int num_samples = num_batches * num_spatial;
  int num_blocks = (num_samples - 1) / num_threads + 1;
  cuda::MergePredictions<<<num_blocks, num_threads>>>(out.dptr_, cls_prob.dptr_,
    box_pred.dptr_, anchors.dptr_, num_classes, num_spatial, num_batches,
    threshold, clip);
  GRID_ANCHOR_DETECTION_CUDA_CHECK(hipPeekAtLastError());
}

template<typename DType>
inline void GridAnchorNonMaximumSuppression(const Tensor<gpu, 3, DType> &out,
                                  const Tensor<gpu, 3, DType> &temp_space,
                                  float nms_threshold, bool force_suppress) {
  int num_anchors = out.size(1);
  int total_threads = num_anchors / 2 + 1;
  const int num_threads = WARPS_PER_BLOCK * THREADS_PER_WARP;
  int num_blocks = (total_threads - 1) / num_threads + 1;
  // sort detection results
  for (int nbatch = 0; nbatch < out.size(0); ++nbatch) {
    DType *src_ptr = out.dptr_ + nbatch * num_anchors * 6;
    DType *dst_ptr = temp_space.dptr_ + nbatch * num_anchors * 6;
    DType *src = src_ptr;
    DType *dst = dst_ptr;
    for (int width = 2; width < (num_anchors << 1); width <<= 1) {
      int slices = (num_anchors - 1) / (total_threads * width) + 1;
      cuda::MergeSortDescend<<<num_blocks, num_threads>>>(src, dst, num_anchors,
        width, slices, 6, 1);
      GRID_ANCHOR_DETECTION_CUDA_CHECK(hipPeekAtLastError());
      src = src == src_ptr? dst_ptr : src_ptr;
      dst = dst == src_ptr? dst_ptr : src_ptr;
    }
  }
  Copy(out, temp_space, temp_space.stream_);
  // apply nms
  num_blocks = (num_anchors - 1) / num_threads + 1;
  for (int nbatch = 0; nbatch < out.size(0); ++nbatch) {
    DType *ptr = out.dptr_ + nbatch * num_anchors * 6;
    for (int pos = 0; pos < num_anchors; ++pos) {
      // suppress against position: pos
      cuda::ApplyNMS<<<num_blocks, num_threads>>>(ptr, pos, num_anchors,
        6, 0, 2, force_suppress, nms_threshold);
      GRID_ANCHOR_DETECTION_CUDA_CHECK(hipPeekAtLastError());
    }
  }
}
}  // namespace mshadow

namespace mxnet {
namespace op {
template<>
Operator *CreateOp<gpu>(GridAnchorDetectionParam param, int dtype) {
  Operator *op = NULL;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new GridAnchorDetectionOp<gpu, DType>(param);
  });
  return op;
}
}  // namespace op
}  // namespace mxnet
